#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>

// Thread block size
#define BLOCK_SIZE 16

// Matrix dimensions
// (chosen as multiples of the thread block size for simplicity)
#define WA (16 * BLOCK_SIZE) // Matrix A width
#define HA (16 * BLOCK_SIZE) // Matrix A height
#define WB (16 * BLOCK_SIZE) // Matrix B width
#define HB WA  // Matrix B height
#define WC WB  // Matrix C width 
#define HC HA  // Matrix C height

//sequential code implemented on cpu
void computeGold(float* C, const float* A, const float* B, unsigned int hA, unsigned int wA, unsigned int wB)
{
    for (unsigned int i = 0; i < hA; ++i)
        for (unsigned int j = 0; j < wB; ++j)
		{
            double sum = 0;
            for (unsigned int k = 0; k < wA; ++k) 
			{
                double a = A[i * wA + k];
                double b = B[k * wB + j];
                sum += a * b;
            }
            C[i * wB + j] = (float)sum;
        }
}

// Initialize a matrix with random float entries.
void randomInit(float* data, int size)
{
    for (int i = 0; i < size; ++i)
        data[i] = rand() / (float)RAND_MAX;
}

//Compare the cpu's result with gpu's 
void printDiff(float *data1, float *data2, int width, int height)
{
  int i,j,k;
  int error_count=0;
  for (j=0; j<height; j++) 
  {
    for (i=0; i<width; i++) 
	{
      k = j*width+i;
      if (data1[k] != data2[k]) 
	  {
         error_count++;
      }
    }
  }
  printf("Total Errors = %d \n", error_count);
}

// matrix multiplication kernel on GPU
__global__ void matrixMul( float* C, float* A, float* B, int wA, int wB)
{
     // Declaration of the shared memory array As used to
    // store the sub-matrix of A
    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

    // Declaration of the shared memory array Bs used to
    // store the sub-matrix of B
    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

	// Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Index of the first sub-matrix of A processed by the block
    int aBegin = wA * BLOCK_SIZE * by;

    // Index of the last sub-matrix of A processed by the block
    int aEnd   = aBegin + wA - 1;

    // Step size used to iterate through the sub-matrices of A
    int aStep  = BLOCK_SIZE;

    // Index of the first sub-matrix of B processed by the block
    int bBegin = BLOCK_SIZE * bx;

    // Step size used to iterate through the sub-matrices of B
    int bStep  = BLOCK_SIZE * wB;

    // Csub is used to store the element of the block sub-matrix
    // that is computed by the thread
    float Csub = 0;

    // Loop over all the sub-matrices of A and B
    // required to compute the block sub-matrix
    for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep) 
	{
        // Load the matrices from device memory
        // to shared memory; each thread loads
        // one element of each matrix
        As[ty][tx] = A[a + wA * ty + tx];
        Bs[ty][tx] = B[b + wB * ty + tx];

        // Synchronize to make sure the matrices are loaded
        __syncthreads();

        // Multiply the two matrices together;
        // each thread computes one element
        // of the block sub-matrix
        for (int k = 0; k < BLOCK_SIZE; ++k)
            Csub += As[ty][k] * Bs[k][tx];

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    // Write the block sub-matrix to device memory;
    // each thread writes one element
    int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
    C[c + wB * ty + tx] = Csub;
}


int main(int argc, char **argv)
{
	// set seed for rand()
    srand((unsigned)time(NULL));

    // allocate host memory for matrices A and B
    unsigned int size_A = WA * HA;
    unsigned int mem_size_A = sizeof(float) * size_A;
    float* h_A = (float*) malloc(mem_size_A);
    unsigned int size_B = WB * HB;
    unsigned int mem_size_B = sizeof(float) * size_B;
    float* h_B = (float*) malloc(mem_size_B);

    // initialize host memory
    randomInit(h_A, size_A);
    randomInit(h_B, size_B);

    // allocate device memory
    float* d_A;
    hipMalloc((void**) &d_A, mem_size_A);
    float* d_B;
    hipMalloc((void**) &d_B, mem_size_B);

    // copy host memory to device
    hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice) ;
    hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice) ;

    // allocate device memory for result
    unsigned int size_C = WC * HC;
    unsigned int mem_size_C = sizeof(float) * size_C;
    float* d_C;
    hipMalloc((void**) &d_C, mem_size_C);

    // allocate host memory for the result
    float* h_C = (float*) malloc(mem_size_C);
    
    // create and start gpu_timer by cuda_event
    float gpu_timer = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // setup execution parameters
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid(WC / threads.x, HC / threads.y);

    // execute the kernel
    hipEventRecord(start, 0);
    matrixMul<<< grid, threads >>>(d_C, d_A, d_B, WA, WB);
    hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

    // stop and destroy gpu_timer
    hipEventElapsedTime(&gpu_timer, start, stop);
    printf("GPU Processing time: %f (s) \n", gpu_timer/1000);

    // copy result from device to host
    hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost) ;

    // compute reference solution
    float* reference = (float*) malloc(mem_size_C);
    double cpu_timer = 0;
    double begin,end;
    begin = clock();
    computeGold(reference, h_A, h_B, HA, WA, WB);
    end = clock();
    cpu_timer = (double)(end - begin)/CLOCKS_PER_SEC;

    // // check result
    // printDiff(reference, h_C, WC, HC);

    // print timers
    printf("CPU Processing time: %f (s) \n", cpu_timer);
    printf("Speedup: %f \n", 1000*cpu_timer/gpu_timer);

    // clean up memory
    free(h_A);
    free(h_B);
    free(h_C);
    free(reference);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipEventDestroy(start);
    hipEventDestroy(stop);

}